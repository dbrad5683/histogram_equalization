#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
extern "C" {
    #include "jpegreadwrite.h"
}

static const int NUM_BINS = 256;

__global__ void histogram(unsigned char *data, unsigned int *buf, int size, int NUM_PARTS) {

    // Pixel coordinates
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Linear thread index within a block
    int t = threadIdx.x + (threadIdx.y * blockDim.x); 

    // Linear block index within a grid
    int b = blockIdx.x + (blockIdx.y * gridDim.x);

    // Absolute linear thread index
    int i = x + (y * blockDim.x * gridDim.x);

    // Initialize temporary accumulation array in global memory
    unsigned int *my_buf = buf + (b * NUM_BINS);
    my_buf[t] = 0;

    // Count NUM_PARTS elements per thread for histogram
    int step = (int)ceil((float)size / (float)NUM_PARTS);
    if (i < step) {
        for (int p = 0; p < NUM_PARTS; p++) {
            int idx = i + (p * step);
            if (idx < size) {
                atomicAdd(&my_buf[data[idx]], 1);
            }
        }
    }
}

__global__ void accumulate(unsigned int *buf, int NUM_BLOCKS) {

    int i = threadIdx.x;
    unsigned int total = 0;

    // Accumulate partial histograms into single histogram
    for (int j = 0; j < NUM_BLOCKS; j++) {
        total += buf[i + (j * NUM_BINS)];
    }
    __syncthreads();

    buf[i] = total;

}

__global__ void equalize(unsigned int *buf, int size, int cdf_min) {

    // Absolute linear thread index
    int i = threadIdx.x;

    // Equalize
    buf[i] = round((float)(NUM_BINS - 1) * (buf[i] - cdf_min) / (size - cdf_min));

}

__global__ void update(unsigned char *data, unsigned int *buf, int size) {

    // Pixel coordinates
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Absolute linear thread index
    int i = x + (y * blockDim.x * gridDim.x);

    if (i < size) {
        data[i] = (unsigned char)buf[data[i]];
    }
    
}

int scan(unsigned int *buf) {

    // Scan buffer to compute cumulative distribution function (cdf)
    int cdf_min = 0;

    for(int i = 1; i < NUM_BINS; i++) {

        buf[i] += buf[i - 1];

        if ((cdf_min == 0) && (buf[i] > 0)) {
            cdf_min = buf[i];
        }
    }

    return(cdf_min);

}

void my_print(unsigned char *data, int rows, int cols) {

    for(int n = 0; n < 10; n++) {
        for(int m = 0; m < 10; m++) {
            printf("%4hhu", data[m + (n * cols)]);
        }
        printf("\n");
    }
    printf("\n");
}

int test(void) {

    int rows = 4320;
    int cols = 7680;
    int size = rows * cols;

    int NUM_PARTS = 32;
    int NUM_BLOCKS = (int)ceil((float)size / (float)(NUM_BINS * NUM_PARTS));
    dim3 UPDATE_GRID(ceil(sqrt(NUM_BLOCKS * NUM_PARTS)), ceil(sqrt(NUM_BLOCKS * NUM_PARTS)));

    unsigned char *h_data;
    unsigned int *h_buf;
    h_data = (unsigned char *)malloc(size * sizeof *h_data);
    h_buf = (unsigned int*)malloc(NUM_BINS * sizeof *h_buf);

    unsigned char *d_data;
    unsigned int *d_buf;
    hipMalloc(&d_data, size * sizeof *d_data);
    hipMalloc(&d_buf, NUM_BINS * NUM_BLOCKS * sizeof *d_buf);

    // unsigned int *h_buf_test;
    // h_buf_test = (unsigned int *)malloc(NUM_BINS * sizeof *h_buf_test);

    // for (int i = 0; i < NUM_BINS; i++) {
    //     h_buf_test[i] = 0;
    // }

    for (int i = 0; i < size; i++) {
        h_data[i] = (unsigned char)((rand() % 75) + 150);
        // h_buf_test[h_data[i]] += 1;
    }

    my_print(h_data, rows, cols);

    hipMemcpy(d_data, h_data, size * sizeof *h_data, hipMemcpyHostToDevice);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    histogram<<<NUM_BLOCKS, NUM_BINS>>>(d_data, d_buf, size, NUM_PARTS);
    accumulate<<<1, NUM_BINS>>>(d_buf, NUM_BLOCKS);

    hipMemcpy(h_buf, d_buf, NUM_BINS * sizeof *d_buf, hipMemcpyDeviceToHost);
    int cdf_min = scan(h_buf);
    hipMemcpy(d_buf, h_buf, NUM_BINS * sizeof *h_buf, hipMemcpyHostToDevice);

    equalize<<<1, NUM_BINS>>>(d_buf, size, cdf_min);
    update<<<UPDATE_GRID, NUM_BINS>>>(d_data, d_buf, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(h_data, d_data, size * sizeof *h_data, hipMemcpyDeviceToHost);
    // hipMemcpy(h_buf, d_buf, NUM_BINS * sizeof *h_buf, hipMemcpyDeviceToHost);

    // for (int i = 0; i < NUM_BINS; i++) {
    //     unsigned int total = 0;
    //     for (int j = 0; j < NUM_BLOCKS; j++) {
    //         total += h_buf[i + (j * NUM_BINS)];
    //     }
    //     h_buf[i] = total;
    // }

    my_print(h_data, rows, cols);

    // for (int i = 0; i < NUM_BINS; i++) {
    //     printf("%4d, %5u, %5u, %5u\n", i, h_buf[i], h_buf_test[i], h_buf[i] - h_buf_test[i]);
    // }

    // printf("%d\n", cdf_min);
    // printf("%f\n", time);

    hipFree(d_data);
    hipFree(d_buf);
    free(h_data);
    free(h_buf);

    return(0);

}

int main(int argc, char **argv) {

    if(argc < 2) {
        fprintf(stderr, "Need jpeg file\n");
        return(-1);
    }

    char *filename = argv[1];

    // Read jpeg
    struct Image img;
    jpegread(filename, &img);

    int rows = img.height;
    int cols = img.width;
    int size = rows * cols;

    int NUM_PARTS = 256;
    int NUM_BLOCKS = (int)ceil((float)size / (float)(NUM_BINS * NUM_PARTS));
    dim3 UPDATE_GRID(ceil(sqrt(NUM_BLOCKS * NUM_PARTS)), ceil(sqrt(NUM_BLOCKS * NUM_PARTS)));

    unsigned char *h_data;
    unsigned int *h_buf;
    h_data = (unsigned char *)malloc(size * sizeof *h_data);
    h_buf = (unsigned int *)malloc(NUM_BINS * sizeof *h_buf);

    unsigned char *d_data;
    unsigned int *d_buf;
    hipMalloc(&d_data, size * sizeof *d_data);
    hipMalloc(&d_buf, NUM_BINS * NUM_BLOCKS * sizeof *d_buf);

    // Get Y component
    for(int j = 0; j < rows; j++) {
        for(int i = 0; i < cols; i++) {
            int offset = (j * cols * 3) + (i * 3);
            h_data[i + (j * cols)] = (unsigned char)img.buffer[offset];
        }
    }

    // my_print(h_data, rows, cols);

    hipMemcpy(d_data, h_data, size * sizeof *h_data, hipMemcpyHostToDevice);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    histogram<<<NUM_BLOCKS, NUM_BINS>>>(d_data, d_buf, size, NUM_PARTS);
    accumulate<<<1, NUM_BINS>>>(d_buf, NUM_BLOCKS);

    hipMemcpy(h_buf, d_buf, NUM_BINS * sizeof *d_buf, hipMemcpyDeviceToHost);
    int cdf_min = scan(h_buf);
    hipMemcpy(d_buf, h_buf, NUM_BINS * sizeof *h_buf, hipMemcpyHostToDevice);

    equalize<<<1, NUM_BINS>>>(d_buf, size, cdf_min);
    update<<<UPDATE_GRID, NUM_BINS>>>(d_data, d_buf, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(h_data, d_data, size * sizeof *h_data, hipMemcpyDeviceToHost);

    // my_print(h_data, rows, cols);

    printf("%f\n", time);

    // Update Y component
    for(int j = 0; j < rows; j++) {
        for(int i = 0; i < cols; i++) {
            int offset = (j * cols * 3) + (i * 3);
            img.buffer[offset] = h_data[i + (j * cols)];
        }
    }

    // Write jpeg
    jpegwrite("out.jpg", &img, 100);

}
